#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

enum algo_impl {
  IMPL_HOST,
  IMPL_GPU_SIMPLE,
  IMPL_GPU_TILED,
};

enum {
  TILE_SIZE = 8,
};

static void simple_mmult(float *A, float *B, float *C, size_t N) {
  for (size_t i = 0; i < N; i++) {
    for (size_t j = 0; j < N; j++) {
      C[i * N + j] = 0;
      for (size_t k = 0; k < N; k++) {
        C[i * N + j] += A[i * N + k] * B[k * N + j];
      }
    }
  }
}

__global__ void mmult_gpu_simple(float *A, float *B, float *C, size_t N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  float val = 0.0f;

  for (size_t k = 0; k < N; k++) {
    val += A[N * i + k] * B[N * k + j];
  }

  C[N * i + j] = val;
}

__global__ void mmult_gpu_tiled(float *A, float *B, float *C, size_t N) {
  __shared__ float As[TILE_SIZE][TILE_SIZE];
  __shared__ float Bs[TILE_SIZE][TILE_SIZE];

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int row = blockIdx.y * TILE_SIZE + ty;
  int col = blockIdx.x * TILE_SIZE + tx;

  float val = 0.0f;

  for (size_t k = 0; k < N / TILE_SIZE; k++) {
    As[ty][tx] = A[row * N + (k * TILE_SIZE + tx)];
    Bs[ty][tx] = B[(k * TILE_SIZE + ty) * N + col];

    __syncthreads();

    for (int m = 0; m < TILE_SIZE; m++) {
      val += As[ty][m] * Bs[m][tx];
    }
    //__syncthreads();
  }
  C[N * row + col] = val;
}

static void mrand(float *A, size_t size) {
  for (int i = 0; i < size * size; i++) {
    A[i] = (float)(rand() % 1000);
  }
}

static void mdump(float *A, size_t size, const char *fname) {
  FILE *f = NULL;
  f = fopen(fname, "wb");
  if (!f) {
    perror("fopen");
    exit(-1);
  }

  for (size_t row = 0; row < size; row++) {
    for (size_t col = 0; col < size - 1; col++) {
      fprintf(f, "%4.4f,", A[row * size + col]);
    }
    fprintf(f, "%4.4f", A[row * size + size - 1]);
    fprintf(f, "\n");
  }

  if (f) {
    fclose(f);
  }
}

int main(int argc, char **argv) {
  enum algo_impl algo = IMPL_GPU_TILED;
  unsigned matrix_size = 32;

  if (argc < 2) {
    printf("Usage: %s [tiled|simple|cpu] size\n", argv[0]);
    return -1;
  }

  if (!strcmp(argv[1], "tiled")) {
    algo = IMPL_GPU_TILED;
  } else if (!strcmp(argv[1], "simple")) {
    algo = IMPL_GPU_SIMPLE;
  } else if (!strcmp(argv[1], "cpu")) {
    algo = IMPL_HOST;
  }

  if (argc > 2) {
    sscanf(argv[1], "%u", &matrix_size);
    if (matrix_size & (matrix_size - 1)) {
      fprintf(stderr, "matrix size '%u' is not power of two\n", matrix_size);
      exit(-1);
    }
  }

  srand(time(NULL));

  size_t alloc_size = matrix_size * matrix_size * sizeof(float);

  float *host_A = (float *)malloc(alloc_size);
  float *host_B = (float *)malloc(alloc_size);
  float *host_C = (float *)malloc(alloc_size);

  float *gpu_A = NULL, *gpu_B = NULL, *gpu_C = NULL;

  hipMalloc((void **)&gpu_A, alloc_size);
  hipMalloc((void **)&gpu_B, alloc_size);
  hipMalloc((void **)&gpu_C, alloc_size);

  dim3 block(TILE_SIZE, TILE_SIZE, 1);
  dim3 grid(matrix_size / block.x, matrix_size / block.y);

  if (!host_A || !host_B || !host_C || !gpu_A || !gpu_B || !gpu_C) {
    return -1;
  }

  mrand(host_A, matrix_size);
  mrand(host_B, matrix_size);
  mdump(host_A, matrix_size, "mtx_a.csv");
  mdump(host_B, matrix_size, "mtx_b.csv");

  hipMemcpy(gpu_A, host_A, alloc_size, hipMemcpyHostToDevice);
  hipMemcpy(gpu_B, host_B, alloc_size, hipMemcpyHostToDevice);

  switch (algo) {
  case IMPL_GPU_TILED:
    mmult_gpu_tiled << <grid, block>>> (gpu_A, gpu_B, gpu_C, matrix_size);
    hipMemcpy(host_C, gpu_C, alloc_size, hipMemcpyDeviceToHost);
    break;
  case IMPL_GPU_SIMPLE:
    mmult_gpu_simple << <grid, block>>> (gpu_A, gpu_B, gpu_C, matrix_size);
    hipMemcpy(host_C, gpu_C, alloc_size, hipMemcpyDeviceToHost);
    break;
  case IMPL_HOST:
    // memset(host_C, 0, alloc_size);
    simple_mmult(host_A, host_B, host_C, matrix_size);
    break;
  }

  mdump(host_C, matrix_size, "mtx_c.csv");
  hipFree(gpu_A);
  hipFree(gpu_B);
  hipFree(gpu_B);
  free(host_A);
  free(host_B);
  free(host_C);
}
